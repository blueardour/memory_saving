#include "hip/hip_runtime.h"
/*
 * Cuda kernels for quantization
 */

#include <torch/extension.h>
#include <ATen/CUDAGeneratorImpl.h>
#include <THC/THCAtomics.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCK_Y_DIM_MAX ((((int64_t)(1)) << 16) - 1)
#define fmax(a, b) ((a) > (b) ? (a): (b))
// #define fmin(a, b) ((a) < (b) ? (a): (b))

using torch::IntArrayRef;
using torch::Tensor;


// Pack float16/32 data into int8 bit stream
template<typename scalar_t, bool boundary_check>
__global__ void pack_single_precision_kernel(int32_t bits,
                                             const scalar_t* __restrict__ data,
                                             const scalar_t* __restrict__ scale,
                                             const scalar_t* __restrict__ shift,
                                             int8_t* __restrict__ packed
//                                              std::pair<uint64_t, uint64_t> seeds
                                            ) {
  const int64_t global_thread_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

//   hiprandStatePhilox4_32_10_t state;
//   hiprand_init(seeds.first, global_thread_id, seeds.second, &state);
//   const float noise = hiprand_uniform(&state);

  const int64_t id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

//   previous
  uint8_t local_packed = 0;
  const int32_t val = __float2int_rn(fmaxf((data[id] - shift[threadIdx.x]), 0.0f));
//   const int32_t val = __float2int_rn(fmax((data[id] - shift[threadIdx.x]) * scale[threadIdx.x] + noise - 0.5, 0.0f));
//   const int32_t val = __float2int_rn(fmaxf((data[id] - shift[threadIdx.x]) * scale[threadIdx.x], 0.0f));
  local_packed |= val;
  packed[global_thread_id] = local_packed;
}

// Pack float16/32 data into int8 bit stream
Tensor pack_single_precision_cuda(Tensor data,
                                  Tensor scale,
                                  Tensor shift,
                                  int bits,
                                  bool stochastic) {
  int64_t B = data.size(0);
  int64_t N = data.size(1);
  int64_t C = data.size(2);

  // Compute total bits
  TORCH_CHECK(8 % bits == 0);

  int64_t total_bits = (int64_t)bits * (B * N * C);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(data.device());
  Tensor packed = torch::empty({(total_bits + 8) / 8,}, options);

//   // Random number generator
//   auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
//   std::pair<uint64_t, uint64_t> rng_engine_inputs;
//   {
//     // See Note [Acquire lock when using random generators]
//     std::lock_guard<std::mutex> lock(gen->mutex_);
//     rng_engine_inputs = gen->philox_engine_inputs(C);
//   }
//   TORCH_CHECK(stochastic);

  dim3 block_dim(N, B, 1);
  dim3 thread_dim(C, 1, 1);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_single_precision", ([&] {
    pack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
      bits,
      data.data_ptr<scalar_t>(),
      scale.data_ptr<scalar_t>(), shift.data_ptr<scalar_t>(),
      packed.data_ptr<int8_t>()
//       rng_engine_inputs
      );
  }));
  // int64_t needed_blocks_y = N/group_size;
  // // Call pack kernels
  // for (int64_t block_idx_y_base = 0; block_idx_y_base < needed_blocks_y; block_idx_y_base += BLOCK_Y_DIM_MAX) {
  //   dim3 block_dim(N, std::min(needed_blocks_y - block_idx_y_base, BLOCK_Y_DIM_MAX), 1);
  //   dim3 thread_dim(group_size, 1, 1);

  //   AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_single_precision", ([&] {
  //     pack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
  //       bits,
  //       data.data_ptr<scalar_t>(),
  //       scale.data_ptr<scalar_t>(), shift.data_ptr<scalar_t>(),
  //       packed.data_ptr<int8_t>(),
  //       // rng_engine_inputs,
  //       N, num_groups, group_size, block_idx_y_base);
  //   }));
  // }

  return packed;
}

// Unpack int32 bit stream to float16/32 data
template<typename scalar_t, bool boundary_check>
__global__ void unpack_single_precision_kernel(int32_t bits,
                                               const int8_t* __restrict__ data,
                                               const scalar_t* __restrict__ scale,
                                               const scalar_t* __restrict__ shift,
                                               scalar_t* __restrict__ unpacked) {
  // const int64_t no = blockIdx.y + block_idx_y_base;
  // const int group_id = blockIdx.x;
  // const int d = threadIdx.x;
  // const int64_t global_thread_id = (no * num_groups + group_id) * group_size + d;
  const int64_t global_thread_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

  uint8_t local_packed = data[global_thread_id];
  int mask = ((1 << bits) - 1);
  const int val = local_packed & mask;
  const int64_t id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
//   unpacked[id] = ((scalar_t)val) / scale[threadIdx.x] + shift[threadIdx.x];
  unpacked[id] = ((scalar_t)val) + shift[threadIdx.x];
}

// Unpack int32 bit stream to float16/32 data
Tensor unpack_single_precision_cuda(Tensor data,
                                    int bits,
                                    Tensor scale,
                                    Tensor shift,
                                    int64_t B,
                                    int64_t N,
                                    int64_t C) {
  auto options = torch::TensorOptions().dtype(scale.dtype()).device(data.device());
  Tensor unpacked = torch::empty({B, N, C}, options);

  // dim3 block_dim(num_groups, std::min(needed_blocks_y - block_idx_y_base, BLOCK_Y_DIM_MAX), 1);
  // dim3 thread_dim(group_size, 1, 1);
  dim3 block_dim(N, B, 1);
  dim3 thread_dim(C, 1, 1);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "unpack_single_precision", ([&] {
    unpack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
      bits,
      data.data_ptr<int8_t>(),
      scale.data_ptr<scalar_t>(), shift.data_ptr<scalar_t>(),
      unpacked.data_ptr<scalar_t>()
      );
  }));

  // int64_t needed_blocks_y = N/group_size;
  // for (int64_t block_idx_y_base = 0; block_idx_y_base < needed_blocks_y; block_idx_y_base += BLOCK_Y_DIM_MAX) {
  //   dim3 block_dim(num_groups, std::min(needed_blocks_y - block_idx_y_base, BLOCK_Y_DIM_MAX), 1);
  //   dim3 thread_dim(group_size, 1, 1);

  //   AT_DISPATCH_FLOATING_TYPES_AND_HALF(scale.scalar_type(), "unpack_single_precision", ([&] {
  //     unpack_single_precision_kernel<scalar_t, false><<<block_dim, thread_dim>>>(
  //       bits,
  //       data.data_ptr<int8_t>(),
  //       scale.data_ptr<scalar_t>(), shift.data_ptr<scalar_t>(),
  //       unpacked.data_ptr<scalar_t>(),
  //       N, num_groups, group_size, block_idx_y_base);
  //   }));
  // }

  return unpacked;
}
